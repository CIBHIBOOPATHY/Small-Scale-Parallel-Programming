#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<string.h>
#include<float.h>
#include<math.h>

// Structure to represent the matrix input in triplet format
typedef struct {
    int row;
    int col;
    double val;
} Triplet;

// Structure to represent an ELLPACK matrix
typedef struct {
    int *row_size;
    int *col_idx;
    double *val;

} ELLPACKMatrix;

//Matrix information are initialised
int nrow,ncol,nnz,max_nnz;

//Structure creation for matrix data in triplet and ELLPACK formats
Triplet *triplets;
ELLPACKMatrix ell;


/*
 * Function: compare
 * -----------------
 *   Compares row indices of triplet data
 *
 *   returns: 0 if row indices of triplets x and y match;
 *            1 if row index of triplet x greater than row index of y
 *           -1 if row index of triplet x lesser than row index of y
 */
int compare(const void *x, const void *y)
{
    Triplet *tripletsx=(Triplet*)x;
    Triplet *tripletsy=(Triplet*)y;

    if(tripletsx->row > tripletsy->row)
    {
        return 1;
    }
	else if(tripletsx->row < tripletsy->row)
    {
        return -1;
    }
	else
	{
        return 0;
    }
}

/*
 * Function: read_matrix
 * ---------------------
 *   Reads the matrix market file and processes the type of matrix
 *   and stores the data in a triplet structure
 *   
 *   filename: Name of matrix market file to be read
 *   
 *   returns: None
 */
 void read_matrix(char* filename)
{
    char line[512];
    FILE* file=fopen(filename,"r");

    // Read the first line to identify the type of matrix
    fgets(line,sizeof(line),file);

    char *token;
    const char delimiter[2]=" ";

    //get the fourth and fifth word
    token = strtok(line,delimiter);

	int k=0;
    
    int mtype=0; // if 0 - real ; 1 - pattern;
    int mformat=0; // if 0 - general; if 1 - symmetric;
    while(token != NULL)
    {
        k++;
        if(k == 4)
        {
            if(strncmp(token, "pattern", 7) == 0){
                mtype=1;
            }
        } 
        else if(k == 5) 
        {
            if(strncmp(token, "symmetric", 9) == 0){
                mformat=1;
            }

        }
        token = strtok(NULL, delimiter);
    }
	//Read comments
    do{
        fgets(line,sizeof(line),file);
    }while(line[0]=='%');

    // Read basic matrix information
	sscanf(line,"%d %d %d",&nrow,&ncol,&nnz);

    //memory allocation for triplet storage
    if(mformat==1)
    {
        triplets=(Triplet*)malloc(nnz*2*sizeof(Triplet));
	}
    else
	{
        triplets=(Triplet*)malloc(nnz*sizeof(Triplet));
    }


    //Pattern matrix check and read the data
    if(mtype==1)
    {
        for(int i=0;i<nnz;i++)
        {
            fgets(line,sizeof(line),file);
            sscanf(line,"%d %d",&triplets[i].row,&triplets[i].col);
            triplets[i].val=1.0; //store 1.0 for all (row,col) values
            triplets[i].row--; // correction of row index from matrix file
            triplets[i].col--; // correction of colummn index from matrix file
        }
	}
    else
    {
        for(int i=0;i<nnz;i++)
        {
            fgets(line,sizeof(line),file);
            sscanf(line,"%d %d %lf",&triplets[i].row,&triplets[i].col,&triplets[i].val);
            triplets[i].row--;
            triplets[i].col--;
        }
    }

    int index=nnz;

    //Symmetric matrix check to reconstruct the full symmetric matrix
    if(mformat==1)
    {
        for(int i=0;i<nnz;i++)
        {
            if(triplets[i].row!=triplets[i].col)
            {
                triplets[index].row=triplets[i].col;
                triplets[index].col=triplets[i].row;
                triplets[index].val=triplets[i].val;
                index+=1;
            }
        }
    }

	nnz=index;
    qsort(triplets,nnz,sizeof(Triplet),compare);
    fclose(file);
}

/*
 * Function: convertToELLPACK
 * --------------------------
 *   Converts the triplet matrix structure to ELLPACK format
 *
 *   returns: None
 *
 */
 void convertToELLPACK()
{
    //Calculation of maximum number of non-zeroe values in a row
    max_nnz = 0;
    int* nnz_count =(int *)calloc(nrow, sizeof(int));

    for (int i = 0; i < nnz; i++)
    {
        nnz_count[triplets[i].row]++;
        max_nnz = (max_nnz > nnz_count[triplets[i].row]?max_nnz:nnz_count[triplets[i].row]);
    }


    // Allocate memory for ELLPACK format

    ell.col_idx = (int*) malloc(nrow * max_nnz * sizeof(int));
    ell.val = (double*) malloc(nrow * max_nnz * sizeof(double));
    ell.row_size = (int*) calloc(nrow,sizeof(int));

    printf("\nMemory allocated\n");

    // Convert triplets to ELLPACK format
    for (int i = 0; i < nnz; i++)
    {
        int row = triplets[i].row;
        int col = triplets[i].col;
        int idx = ell.row_size[row];

        ell.val[row * max_nnz + idx] = triplets[i].val;
        ell.col_idx[row * max_nnz + idx] = col;
        ell.row_size[row]++;
    }

	// Free memory
    free(nnz_count);
}

/*
 * Function: serial_ELLPACK
 * ------------------------
 *   Multiplies the triplet matrix with a vector in serial
 *
 *   *x contains the vector data; *y stores the results obtained during multiplication
 *   returns: None
 *
 */
void serial_ELLPACK(int *x,double *y)
{
    for(int i=0;i<nrow;i++)
    {
        double t=0.0;
        for(int j=0;j<max_nnz;j++)
        {
            int idx=ell.col_idx[i * max_nnz + j];

            if(ell.val[i * max_nnz + j]==0.0)
            {
                continue;
            }
            t+=ell.val[i * max_nnz + j]*x[idx];
        }
        y[i]=t;
    }
}

/*
 * Function: validate
 * ------------------
 *   Validation of parallel multiplication result vector by comparing it with serial
 *   multiplication result vector
 *
 *   *y contains the serial results; *z contains the parallel results
 *
 *   returns: None
 *
 */
void validate(double *y,double *z)
{
    int valid=0; // Stores the number of matched values
    int tol=1000;
    for(int i=0;i<nrow;i++)
    {
        if(fabs(z[i]-y[i])<tol)
        {
            valid++;
        }
    }
	if(valid==nrow)
    {
        printf("\nValidated\n");
    }
	else
	{
        printf("\nNot validated\n");
    }
}

/*
 * Function: parallel_ELLPACK
 * --------------------------
 *   Parallelization of serial_ELLPACK function
 *
 *   returns: None
 *
 */
 __global__ void parallel_ELLPACK(int* col_idx, double* val, int max_nnz, int nrow, int* x, double* z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < nrow)
    {
        double t = 0.0;
        for(int j=0; j<max_nnz; j++)
        {
            int idx = col_idx[i * max_nnz + j];
            if(val[i * max_nnz + j] == 0.0)
            {
                continue;
            }
            t += val[i * max_nnz + j] * x[idx];
        }
        z[i] = t;
    }
}

/*
 * Function: ellpack_matrix_vector_multiply
 * -----------------------------------------
 *   Calls the kernels for serial and parallel matrix vector dot product
 *
 *   returns: None
 *
 */
    // initialise vector to be multiplied with the ellpack matrix
    int* x = (int*)malloc(nrow * sizeof(int));


    // initialise the result vectors of serial and parallel computations
    double* y = (double*)malloc(nrow * sizeof(double)); // Serial result vector
    double* z = (double*)malloc(nrow * sizeof(double)); // Parallel result vector

    // Input data into x vector
    for(int i=0; i<nrow; i++)
    {
        x[i] = i;
    }

	// allocate memory on the device
    int* d_col_idx;
    double* d_val;
    int* d_x;
    double* d_z;

    hipMalloc((void**)&d_col_idx, nrow * max_nnz * sizeof(int));
    hipMalloc((void**)&d_val, nrow * max_nnz * sizeof(double));
    hipMalloc((void**)&d_x, nrow * sizeof(int));
    hipMalloc((void**)&d_z, nrow * sizeof(double));

    // copy matrix data from host to device
    hipMemcpy(d_col_idx, ell.col_idx, nrow * max_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, ell.val, nrow * max_nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, nrow * sizeof(int), hipMemcpyHostToDevice);

    // kernel parameters
    int block_size = 256;
    int num_blocks = (nrow + block_size - 1) / block_size;

    float avg_ptime = 0;
    //launch kernel
    for(int i=0;i<1000;i++)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        parallel_ELLPACK<<<num_blocks, block_size>>>(d_col_idx, d_val, max_nnz, nrow, d_x, d_z);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ptime = 0;
        hipEventElapsedTime(&ptime, start, stop);
        float seconds = ptime / 1000.0f;
        avg_ptime+=seconds;
    }
	printf("\nELLPACK Computational Time for Parallel Kernel:%f seconds\n",avg_ptime/1000);

    // copy results from device to host
    hipMemcpy(z, d_z, nrow * sizeof(double), hipMemcpyDeviceToHost);

    // serial sparse matrix multiplication
    serial_ELLPACK(x, y);

    // Validation of results
    validate(y, z);

    // free memory on the device
    hipFree(d_col_idx);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_z);

    // free memory on the host
    free(x);
    free(y);
    free(z);
}

int main()
{
    //Read matrix
    read_matrix("/scratch/s399510/cage4.mtx");

    convertToELLPACK();
    ellpack_matrix_vector_multiply();

    // Free memory
    free(triplets);
    free(ell.row_size);
    free(ell.col_idx);
    free(ell.val);
    return 0;
}







